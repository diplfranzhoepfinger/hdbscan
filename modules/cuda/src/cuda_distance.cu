#include "hip/hip_runtime.h"
/*
 * cuda_distance.cu
 *
 * Copyright 2019 Onalenna Junior Makhura
 *
 * Permission is hereby granted, free of charge, to any person
 * obtaining a copy of this software and associated documentation files
 * (the "Software"), to deal in the Software without restriction,
 * including without limitation the rights to use, copy, modify, merge,
 * publish, distribute, sublicense, and/or sell copies of the Software,
 * and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT.  IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS
 * BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN
 * ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */
#include "hdbscan/cuda_distance.h"

__device__
uint TRIANGULAR_D(uint n) {
	return (n * n + n) / 2;
}

__device__
uint deviceFistTriangleNum(uint p) {
	return ceilf((sqrtf(8 * p + 1) - 1) / 2);
}

__global__
void distance_compute_kernel(double* a_in, double *d_out, uint w, uint h, uint d) {
	uint i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < d) {
		uint r = deviceFistTriangleNum(i+1);
		uint tmp = TRIANGULAR_D(r);
		uint c = i + r - tmp;
		double sum = 0.0, diff = 0.0;

		for (uint k = 0; k < w; k++) {
			double num1 = a_in[r * w + k];
			double num2 = a_in[c * w + k];
			diff = num1 - num2;
			sum += (diff * diff);
		}

		sum = sqrtf(sum);
		d_out[i] = sum;
	}
}